#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>

template<typename T>
constexpr T constexpr_max(T a, T b) {
    return (a > b) ? a : b;
}

template <int InputChannels, int InputLength, int Padding, int KernelSize, int ChannelsPerThread>
__global__ void conv1d(float *d_input, float *d_weight, float *d_bias, float *d_output)
{
    //define constants
    constexpr int SharedMemLength = constexpr_max(InputLength, KernelSize);
    const int blockId = blockIdx.x;
    const int tdIdx = threadIdx.x;
    const int laneIdx = threadIdx.x % warpSize;
    const int warpIdx = threadIdx.x / warpSize;

    const int input_accesses_per_thread = (InputChannels * InputLength)/(4 * blockDim.x); 
    const int weight_accesses_per_thread = (InputChannels * KernelSize)/(blockDim.x); 
    const int weight_offset = blockId * InputChannels * KernelSize;
    const int padded_input_length = InputLength + Padding * 2;
    const int shared_mem_offset_denom = (InputLength * ChannelsPerThread) < 32 ? 32 : (InputLength * ChannelsPerThread);


    //static mem allocations
    float regInput[padded_input_length*ChannelsPerThread] = {0};
    float regFilter[KernelSize*ChannelsPerThread];
    __shared__ float shared_mem[InputChannels * SharedMemLength];

    //load input from global memory into shared memory 
    for (int channelIndex = 0; channelIndex < input_accesses_per_thread; ++channelIndex){
        int td_offset = 4 * (channelIndex * blockDim.x + tdIdx); 
        int smem_offset = td_offset/shared_mem_offset_denom; 
        float4 data = *reinterpret_cast<float4*>(&d_input[td_offset]);
        shared_mem[td_offset + smem_offset + 0] = data.x; 
        shared_mem[td_offset + smem_offset + 1] = data.y; 
        shared_mem[td_offset + smem_offset + 2] = data.z; 
        shared_mem[td_offset + smem_offset + 3] = data.w; 
    }

    __syncthreads(); 

    //load input from shared memory into thread registers
    for (int channelIndex = 0; channelIndex < ChannelsPerThread; ++channelIndex){
        for (int colIndex = 0; colIndex < InputLength; ++colIndex){
            int regIndex = Padding + channelIndex * padded_input_length + colIndex;
            int sharedMemIndex = InputLength * (ChannelsPerThread * tdIdx + channelIndex) + colIndex;
            int smem_offset = sharedMemIndex/shared_mem_offset_denom; 
            regInput[regIndex] = shared_mem[sharedMemIndex + smem_offset];
        }
    }

    __syncthreads(); 

    //load weights from global memory into shared memory 
    for (int channelIndex = 0; channelIndex < weight_accesses_per_thread; ++channelIndex){
        int td_offset = (channelIndex * blockDim.x) + tdIdx;
        shared_mem[td_offset] = d_weight[td_offset + weight_offset];
    }

    __syncthreads(); 

    //load weights from shared memory to thread registers
    for (int channelIndex = 0; channelIndex < ChannelsPerThread; ++channelIndex){
        for (int colIdx = 0; colIdx < KernelSize; ++colIdx){
            int regIndex = channelIndex * KernelSize + colIdx;
            int sharedMemIndex = KernelSize * (ChannelsPerThread * tdIdx + channelIndex) + colIdx;
            regFilter[regIndex] = shared_mem[sharedMemIndex];
        }
    }

    //outer loop iterates over each element in output vector
    for (int tileIdx = 0; tileIdx < InputLength; ++tileIdx){
        float res = 0.0;
        
        //inner loop performs dot product over all kernel positions and accumulates results
        for(int dotIdx = 0; dotIdx < KernelSize; ++dotIdx){
            for(int channelIndex = 0; channelIndex < ChannelsPerThread; ++channelIndex){
                res += regInput[tileIdx + dotIdx + (channelIndex * padded_input_length)] * regFilter[dotIdx + (channelIndex * KernelSize)];
            }
        }
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            res += __shfl_down_sync(0xffffffff, res, offset);
        }
        
        if (threadIdx.x == 0) {
            atomicAdd(&d_output[blockIdx.x * InputLength + tileIdx], d_bias[blockIdx.x]);
        }

        if (laneIdx == 0) {
            atomicAdd(&d_output[blockIdx.x * InputLength + tileIdx], res);
        }
    }
}

template <int CHUNK_SIZE>
__global__ void parallelGroupNormMishKernel(float* d_input, float* d_output, float* d_weights, float* d_bias, const int input_length) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ float sharedSum[CHUNK_SIZE]; // Shared memory for storing sum and sum of squares

    //1. Load data into shared memory
    float dataValue = d_input[bid * CHUNK_SIZE + tid]; 
    sharedSum[tid] = dataValue;
    __syncthreads();

    //2. Reduction to compute total sum (mean calculation)
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedSum[tid] += sharedSum[tid + stride];
        }
        __syncthreads();
    }

    //3. calculates the square of the difference between the value for the current thread and the mean for the group
    float mean = sharedSum[0] / CHUNK_SIZE;
    float diff = dataValue - mean;
    sharedSum[tid] = diff * diff; //save the squared difference in shared memory
    __syncthreads();

    //4. Reduction to compute total sum of squares (variance calculation)
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedSum[tid] += sharedSum[tid + stride];
        }
        __syncthreads();
    }

    //5. calculates the normalized value
    float variance = sharedSum[0] / CHUNK_SIZE;
    int global_td = bid * CHUNK_SIZE + tid;
    int weight_bias_idx = global_td / input_length;
    float invStdDev = rsqrtf(variance + 1e-5); // Using epsilon = 1e-5
     float normVal = (dataValue - mean) * invStdDev;
    
    //6. scale normalized value by weight and bias
    float weightedNormVal = normVal * d_weights[weight_bias_idx] + d_bias[weight_bias_idx];

    //7. apply Mish activation using CUDA special functions and store the result in global memory
    float mishVal = weightedNormVal * tanhf(log1pf(expf(weightedNormVal)));
    d_output[bid * CHUNK_SIZE + tid] = mishVal;
}

torch::Tensor conv1d_gnm_cuda(
    torch::Tensor& input,
    torch::Tensor& conv1d_weight_tensor,
    torch::Tensor& conv1d_bias_tensor,
    torch::Tensor& gnm_weight_tensor,
    torch::Tensor& gnm_bias_tensor,
    int output_channels,
    int padding,
    int kernel_size,
    const int channelsPerThread
){
    input = input.contiguous(); 
    conv1d_weight_tensor = conv1d_weight_tensor.contiguous();
    conv1d_bias_tensor = conv1d_bias_tensor.contiguous();
    gnm_weight_tensor = gnm_weight_tensor.contiguous();
    gnm_bias_tensor = gnm_bias_tensor.contiguous();

    float* d_input = input.data_ptr<float>();
    float* d_conv1d_weight = conv1d_weight_tensor.data_ptr<float>();
    float* d_conv1d_bias = conv1d_bias_tensor.data_ptr<float>();
    float* d_gnm_weight = gnm_weight_tensor.data_ptr<float>();
    float* d_gnm_bias = gnm_bias_tensor.data_ptr<float>();

    const int input_channels = input.size(1);
    const int input_length = input.size(2);
    const int output_length = input_length - kernel_size + 1 + 2 * padding;
    const int chunk_size = (output_channels * input_length) / 8; 

    const dim3 conv1d_blocks(output_channels, 1, 1);
    
    const dim3 gnm_blocks(8, 1, 1);
    const dim3 gnm_threads(chunk_size, 1, 1);

    auto options = input.options();
    auto conv1d_out = torch::zeros({1, output_channels, output_length}, options);
    auto gnm_out = torch::empty({1, output_channels, output_length}, options);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (input_channels == 2 && input_length == 16) {
        const dim3 conv1d_threads(input_channels / 1, 1, 1);
        conv1d<2, 16, 2, 5, 1><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 512 && input_length == 4) {
        const dim3 conv1d_threads(input_channels / 8, 1, 1);
        conv1d<512, 4, 2, 5, 8><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 256 && input_length == 16) {
        const dim3 conv1d_threads(input_channels / 2, 1, 1);
        conv1d<256, 16, 2, 5, 2><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 512 && input_length == 8) {
        const dim3 conv1d_threads(input_channels / 4, 1, 1);
        conv1d<512, 8, 2, 5, 4><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 1024 && input_length == 4) {
        const dim3 conv1d_threads(input_channels / 4, 1, 1);
        conv1d<1024, 4, 2, 5, 4><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 256 && input_length == 8) {
        const dim3 conv1d_threads(input_channels / 2, 1, 1);
        conv1d<256, 8, 2, 5, 2><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 1024 && input_length == 4) {
        const dim3 conv1d_threads(input_channels / 4, 1, 1);
        conv1d<1024, 8, 2, 5, 4><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    } else if (input_channels == 2048 && input_length == 4) {
        const dim3 conv1d_threads(input_channels / 4, 1, 1);
        conv1d<2048, 4, 2, 5, 4><<<conv1d_blocks, conv1d_threads, 0, stream>>>(d_input, d_conv1d_weight, d_conv1d_bias, conv1d_out.data_ptr<float>());
    }

    hipError_t conv1d_err = hipGetLastError();
    if (conv1d_err != hipSuccess) {
        throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(conv1d_err)));
    }
    
    if (chunk_size == 512){
        parallelGroupNormMishKernel<512><<<gnm_blocks, gnm_threads, 0, stream>>>(conv1d_out.data_ptr<float>(), gnm_out.data_ptr<float>(), d_gnm_weight, d_gnm_bias, output_length);
    }
    else if (chunk_size == 256){
        parallelGroupNormMishKernel<256><<<gnm_blocks, gnm_threads, 0, stream>>>(conv1d_out.data_ptr<float>(), gnm_out.data_ptr<float>(), d_gnm_weight, d_gnm_bias, output_length);
    }
    else if (chunk_size == 1024){
        parallelGroupNormMishKernel<1024><<<gnm_blocks, gnm_threads, 0, stream>>>(conv1d_out.data_ptr<float>(), gnm_out.data_ptr<float>(), d_gnm_weight, d_gnm_bias, output_length);
    }
    else if (chunk_size == 4){
        parallelGroupNormMishKernel<4><<<gnm_blocks, gnm_threads, 0, stream>>>(conv1d_out.data_ptr<float>(), gnm_out.data_ptr<float>(), d_gnm_weight, d_gnm_bias, output_length);
    }

    hipError_t gnm_err = hipGetLastError();
    if (gnm_err != hipSuccess) {
        throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(gnm_err)));
    }
    
    return gnm_out;
}
